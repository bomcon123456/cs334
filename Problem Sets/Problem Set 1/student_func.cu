#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, they use that to
  //calculate a 1D offset
  int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;
  if (y >= numCols || x >= numRows) {
    return;
  }
  int index = numCols*x + y;
  uchar4 color_pixel = rgbaImage[index];
  unsigned char grey_pixel = (unsigned char)(0.299f*color_pixel.x+ 0.587f*color_pixel.y + 0.114f*color_pixel.z);
  greyImage[index] = grey_pixel;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int   blockWidth = 32;
  const dim3 blockSize(blockWidth, blockWidth, 1);
  int   blockX = numRows/blockWidth + 1;
  int   blockY = numCols/blockWidth + 1;
  const dim3 gridSize( blockX, blockY, 1);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
